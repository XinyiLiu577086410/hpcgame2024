#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstring>
#include <cstdint>
#include <cstdio>

// #define DBG
__device__
uint8_t B6S567(uint8_t now, uint8_t around) {
  if(now == 0) {
    if (around == 6) 
      return 1;
    else
      return 0;
  }
  else if(now == 1) {
    if(around == 5 || around == 6 || around ==7)
      return 1;
    else 
      return 0;
  }
}

__device__ 
void coord_mod(int M, int &x, int &y, int &z) {
  x += M;
  y += M;
  z += M;
  x %= M;
  y %= M;
  z %= M;
}

__device__ 
int coord_map(int M, int x, int y, int z) {
  coord_mod(M, x, y, z);
  return M * M * x + M * y + z;
}

__global__ 
void gameKernel(uint8_t* dst, const uint8_t* src, int M, int* dirx, int* diry, int* dirz) {
  int x = blockIdx.y;
  int y = blockIdx.z;
  int z = threadIdx.x;
  int tx, ty, tz;
  int cur_dst = coord_map(M, x, y, z);
  dst[cur_dst] = 0;
  for(int i=0; i<26; ++i) {
    tx = x + dirx[i];
    ty = y + diry[i];
    tz = z + dirz[i];
    int cur_src = coord_map(M, tx, ty, tz);
    dst[cur_dst] += src[cur_src];
  }
  dst[cur_dst] = B6S567(src[cur_dst], dst[cur_dst]);
}

int main(int argc, char *argv[]) {
  if(argc != 4) {
    std::cout << "Usage:" << argv[0] << "<input file> <output file> <number of iterations>";
  }
  std::ifstream input_file(argv[1], std::ios::binary);
  int64_t N = atoll(argv[3]), M, T;
  input_file.read(reinterpret_cast<char *>(&M), 8);
  input_file.read(reinterpret_cast<char *>(&T), 8);
  int space_size = M * M * M * (sizeof(uint8_t));
  uint8_t * buffer = (uint8_t *) malloc(2 * space_size);
  memset(buffer, 0, space_size*2);
  uint8_t * d_buffer;
  hipMalloc(reinterpret_cast<void**>(&d_buffer), 2 * space_size);
  input_file.read(reinterpret_cast<char*>(buffer), space_size);
  hipMemcpy(d_buffer, buffer, 2 * space_size, hipMemcpyHostToDevice);
  int dirx[26], diry[26], dirz[26], cur = -1, *d_dirx, *d_diry, *d_dirz;
  for(int dx = -1; dx <= 1; dx++) {
    for(int dy = -1; dy <= 1; dy++) {
      for(int dz = -1; dz <= 1; dz++) {
        if(dx|dy|dz) ++cur; else continue;
        dirx[cur] = dx;
        diry[cur] = dy;
        dirz[cur] = dz;
      }
    } 
  }

  hipMalloc((void **)&d_dirx, 26 * sizeof(int));
  hipMalloc((void **)&d_diry, 26 * sizeof(int));
  hipMalloc((void **)&d_dirz, 26 * sizeof(int));
  hipMemcpy(d_dirx, dirx, 26 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_diry, diry, 26 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dirz, dirz, 26 * sizeof(int), hipMemcpyHostToDevice);
  for(int i=0; i<N; ++i){
    if(i % 2 == 0)
      gameKernel<<<dim3(1,M,M),dim3(M,1,1)>>>(d_buffer+space_size, d_buffer, M, d_dirx, d_diry, d_dirz);
    else
      gameKernel<<<dim3(1,M,M),dim3(M,1,1)>>>(d_buffer, d_buffer+space_size, M, d_dirx, d_diry, d_dirz);
    hipDeviceSynchronize();
  }
  hipMemcpy(buffer, d_buffer, 2 * space_size, hipMemcpyDeviceToHost);
  std::ofstream output_file(argv[2], std::ios::binary);
  output_file.write(reinterpret_cast<const char *>(&M), 8);
  output_file.write(reinterpret_cast<const char *>(&N), 8);
  if(N % 2) 
    output_file.write((const char*)buffer, space_size);
  else
    output_file.write((const char*)(buffer+space_size), space_size);

  free(buffer);
  hipFree(d_buffer);
  hipFree(d_dirx);
  hipFree(d_diry);
  hipFree(d_dirz);
  input_file.close();
  output_file.close(); 
  }