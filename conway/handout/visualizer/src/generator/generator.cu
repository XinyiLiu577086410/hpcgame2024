#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <utility>

#include <hip/hip_runtime.h>

#include "world.hpp"

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))
#define BYTES_PER_THREAD 1

#define CUDA_CHECK(cmd) do { \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        printf("[ERROR] CUDA error %s:%d '%s' : %s\n", __FILE__, __LINE__, #cmd, hipGetErrorString(result)); \
        exit(-1); \
    } \
} while(0)

inline void syncAndCheck(const char* const file, int const line, bool force_check = false) {
#ifdef DEBUG
    force_check = true;
#endif
    if (force_check) {
        hipDeviceSynchronize();
        hipError_t result = hipGetLastError();
        if (result) {
            throw std::runtime_error(std::string("[ST] CUDA runtime error: ") + hipGetErrorString(result) + " "
                                    + file + ":" + std::to_string(line) + " \n");
        }
    }
}

#define sync_check_cuda_error() syncAndCheck(__FILE__, __LINE__, false)
#define sync_check_cuda_error_force() syncAndCheck(__FILE__, __LINE__, true)

namespace hpcgame {

void update_world_cuda(World3D& old_world, World3D& new_world, const size_t step) {
  throw std::runtime_error("CUDA not implemented");
}

}